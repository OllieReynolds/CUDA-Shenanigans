#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdlib>

// The device kernel
__global__ void my_first_kernel(float *x) {
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	x[tid] = (float)threadIdx.x;
}

int main() {
	// Setup
	float *hx, *dx;
	int blocks = 2;
	int threads = 8;
	int size = blocks*threads;

	// Allocate host and device memory
	hx = (float*)malloc(size * sizeof(float));
	hipMalloc((void**)&dx, size * sizeof(float));

	// Execute kernel 
	my_first_kernel << <blocks, threads >> > (dx);

	// Copy device memory back to host memory
	hipMemcpy(hx, dx, size * sizeof(float), hipMemcpyDeviceToHost);

	// Output results
	for (int i = 0; i < size; i++) {
		printf(" n, x = %d %f\n", i, hx[i]);
	}

	// Free memory
	hipFree(dx);
	free(hx);
}